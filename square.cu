#include "hip/hip_runtime.h"
#include <stdio.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

__global__
void squareKernel(int *data, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < N)
	{
		data[i] = i * i;
	}
}

int main(int argc, char **argv)
{
	int *h_data;
	int *d_data;
	int sum = 0;

	hipHostAlloc(&h_data, 1000 * sizeof(int), hipHostMallocPortable);
	hipMalloc(&d_data, 1000 * sizeof(int));

	for(int i=0;i<1000;i++)
	{
		h_data[i] = i*i;
	}

	dim3 block(512);
	dim3 grid((1000 + block.x - 1) / block.x);

	hipMemcpy(d_data, h_data, 1000 * sizeof(int), hipMemcpyHostToDevice);
	squareKernel<<<grid, block>>>(d_data, 1000);
	hipMemcpy(h_data, d_data, 1000 * sizeof(int), hipMemcpyDeviceToHost);

	for(int j=0;j<1000;j++)
	{
		sum = sum + h_data[j];
	}
	
	printf("h_data[998] = %d\n", h_data[998]);
	printf("h_data[999] = %d\n", h_data[999]);
	printf("sum = %d\n", sum);
	return 0;
}